#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#define THREAD_PER_BLOCK 256
// latency: 0.656ms
template <int blockSize>
__device__ void BlockReduce(float* sdata) {
  if (blockSize >= 1024) {
    if (threadIdx.x < 512) {
      sdata[threadIdx.x] += sdata[threadIdx.x + 512];
    }
    __syncthreads();
  }
  if (blockSize >= 512) {
    if (threadIdx.x < 256) {
      sdata[threadIdx.x] += sdata[threadIdx.x + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (threadIdx.x < 128) {
      sdata[threadIdx.x] += sdata[threadIdx.x + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (threadIdx.x < 64) {
      sdata[threadIdx.x] += sdata[threadIdx.x + 64];
    }
    __syncthreads();
  }
  // the final warp
  if (threadIdx.x < 32) {
    volatile float* vshm = sdata;
    if (blockDim.x >= 64) {
      vshm[threadIdx.x] += vshm[threadIdx.x + 32];
    }
    vshm[threadIdx.x] += vshm[threadIdx.x + 16];
    vshm[threadIdx.x] += vshm[threadIdx.x + 8];
    vshm[threadIdx.x] += vshm[threadIdx.x + 4];
    vshm[threadIdx.x] += vshm[threadIdx.x + 2]; 
    vshm[threadIdx.x] += vshm[threadIdx.x + 1];
  }
}

template <int blockSize>
__global__ void reduce_v5(float *d_in, float *d_out){
    __shared__ float sdata[THREAD_PER_BLOCK];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    // load: 每个线程加载两个元素到shared mem对应位置
    sdata[tid] = d_in[i] + d_in[i + blockDim.x];
    __syncthreads();
    // compute: reduce in shared mem
    BlockReduce<blockSize>(sdata);

    // store: write back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    //const int N = 32 * 1024 * 1024;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize / 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v5<blockSize / 2><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("resPerBlock : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v5 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}