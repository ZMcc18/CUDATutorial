#include <stdio.h>
#include <hip/hip_runtime.h>


typedef float FLOAT;

/* CUDA kernel function */
__global__ void vec_add(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    /* 2D grid   实际上是2d的grid和1d的block，
gridDim.x表示的block在2dgrid上的x方向上维度，所以blockIdx.y * gridDim.x可以看作是直到你所在的block前所有行的block，之后加上blockIdx.x
就相当于是block in grid,之后再乘blockDim.x相当于是除了当前thread所在的block之前的所有block中的线程数量，最后加上当前的block中的线程索引就是全局线程索引了*/    
    int idx = (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x);
    /* 1D grid */
    // int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) z[idx] = y[idx] + x[idx];
}

void vec_add_cpu(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    for (int i = 0; i < N; i++) z[i] = y[i] + x[i];
}

int main()
{
    int N = 10000;
    int nbytes = N * sizeof(FLOAT);

    /* 1D block */
    int bs = 256;

    /* 2D grid */
    int s = ceil(sqrt((N + bs - 1.) / bs));
    dim3 grid(s, s);
    /* 1D grid */
    // int s = ceil((N + bs - 1.) / bs);
    // dim3 grid(s);

    FLOAT *dx, *hx;
    FLOAT *dy, *hy;
    FLOAT *dz, *hz;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);
    
    /* init time */
    float milliseconds = 0;

    /* alllocate CPU mem */
    hx = (FLOAT *) malloc(nbytes);
    hy = (FLOAT *) malloc(nbytes);
    hz = (FLOAT *) malloc(nbytes);

    /* init */
    for (int i = 0; i < N; i++) {
        hx[i] = 1;
        hy[i] = 1;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    /* launch GPU kernel */
    vec_add<<<grid, bs>>>(dx, dy, dz, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);  


	/* copy GPU result to CPU */
    hipMemcpy(hz, dz, nbytes, hipMemcpyDeviceToHost);

    /* CPU compute */
    FLOAT* hz_cpu_res = (FLOAT *) malloc(nbytes);
    vec_add_cpu(hx, hy, hz_cpu_res, N);

    /* check GPU result with CPU*/
    for (int i = 0; i < N; ++i) {
        if (fabs(hz_cpu_res[i] - hz[i]) > 1e-6) {
            printf("Result verification failed at element index %d!\n", i);
        }
    }
    printf("Result right\n");
    printf("Mem BW= %f (GB/sec)\n", (float)N*4/milliseconds/1e6);///1.78gb/s
    hipFree(dx);
    hipFree(dy);
    hipFree(dz);

    free(hx);
    free(hy);
    free(hz);
    free(hz_cpu_res);
//需要销毁cuda事件 没写上去

// 销毁 CUDA 事件
hipEventDestroy(start);
hipEventDestroy(stop);
    return 0;
}
